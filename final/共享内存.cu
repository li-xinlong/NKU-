#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <string>
#include <fstream>
#include <sstream>
#include <vector>
#include <algorithm>
#include <Windows.h>

using namespace std;

__global__ void FindKernel(unsigned int* A, unsigned int a, int size, bool* found) {
	extern __shared__ unsigned int shared_A[];
	int tid = threadIdx.x;
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < size) {
		shared_A[tid] = A[i];
	}
	__syncthreads();

	if (tid < size && shared_A[tid] == a) {
		*found = true;
	}
}

bool Find(unsigned int* d_A, unsigned int a, int size) {
	bool* d_found;
	bool h_found = false;
	hipMalloc((void**)&d_found, sizeof(bool));
	hipMemcpy(d_found, &h_found, sizeof(bool), hipMemcpyHostToDevice);

	int threadsPerBlock = 128;
	/*int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;*/
	int blocksPerGrid = 8;
	FindKernel << <blocksPerGrid, threadsPerBlock, threadsPerBlock * sizeof(unsigned int) >> > (d_A, a, size, d_found);

	hipDeviceSynchronize();
	hipMemcpy(&h_found, d_found, sizeof(bool), hipMemcpyDeviceToHost);
	hipFree(d_found);

	hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		std::cerr << "CUDA error: " << hipGetErrorString(error) << std::endl;
		return false;
	}

	return h_found;
}

int main() {
	cout << "ExpIndex索引规模为1757" << endl;
	cout << "ExpIndex中最大文档ID为25205174,最小文档ID为0" << endl;
	cout << "ExpQuery最大查询规模为1000" << endl;
	int QueryNum;
	cout << "请输入问题规模(0-1000内的整型)：" << endl;
	cin >> QueryNum;

	LARGE_INTEGER frequency, start, end, end_f;
	double interval;
	QueryPerformanceFrequency(&frequency);
	QueryPerformanceCounter(&start);

	ifstream readIndex("ExpIndex", ios::binary | ios::in);
	if (!readIndex) {
		cerr << "无法打开ExpIndex文件" << endl;
		return 1;
	}

	vector<vector<unsigned int>> index;
	unsigned int n;
	while (readIndex.read(reinterpret_cast<char*>(&n), sizeof(n))) {
		vector<unsigned int> array(n);
		if (!readIndex.read(reinterpret_cast<char*>(array.data()), n * sizeof(unsigned int))) {
			cerr << "读取ExpIndex失败" << endl;
		}
		sort(array.begin(), array.end());
		index.push_back(array);
	}
	readIndex.close();

	ifstream readQuery("ExpQuery", ios::in);
	if (!readQuery) {
		cerr << "无法打开ExpQuery文件" << endl;
		return 1;
	}

	vector<vector<int>> query;
	string line;
	while (getline(readQuery, line)) {
		vector<int> array;
		stringstream readline(line);
		int a;
		while (readline >> a) {
			array.push_back(a);
		}
		sort(array.begin(), array.end());
		query.push_back(array);
	}
	readQuery.close();
	QueryPerformanceCounter(&end_f);
	interval = static_cast<double>(end_f.QuadPart - start.QuadPart) / frequency.QuadPart;
	std::cout << "读取文件时间: " << interval << " 秒" << std::endl;
	cout << "读取文件操作完成" << endl;

	vector<vector<unsigned int>> results;
	for (int i = 0; i < QueryNum; i++) {
		int num = query[i].size();
		vector<unsigned int> result = index[query[i][0]];

		for (int j = 1; j < num; j++) {
			unsigned int* d_index;
			hipMalloc((void**)&d_index, index[query[i][j]].size() * sizeof(unsigned int));
			hipMemcpy(d_index, index[query[i][j]].data(), index[query[i][j]].size() * sizeof(unsigned int), hipMemcpyHostToDevice);

			for (int k = 0; k < result.size(); k++) {
				if (!Find(d_index, result[k], index[query[i][j]].size())) {
					result.erase(result.begin() + k);
					k--;
				}
			}
			hipFree(d_index);
		}
		results.push_back(result);
	}
	QueryPerformanceCounter(&end);
	interval = static_cast<double>(end.QuadPart - start.QuadPart) / frequency.QuadPart;
	std::cout << "代码执行时间: " << interval << " 秒" << std::endl;
	return 0;
}
