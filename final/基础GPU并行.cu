#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include<iostream>
#include <string>
#include <fstream>
#include<sstream>
#include<vector>
#include <algorithm>
#include <Windows.h>
#include <hip/hip_runtime.h>
#include <>

using namespace std;

__global__ void FindKernel(unsigned int* A, unsigned int a, int size, bool* found) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size && A[i] == a) {
		*found = true;
	}
}


bool Find(unsigned int* d_A, unsigned int a, int size) {
	bool* d_found;
	bool h_found = false;
	hipMalloc((void**)&d_found, sizeof(bool));
	hipMemcpy(d_found, &h_found, sizeof(bool), hipMemcpyHostToDevice);

	int threadsPerBlock = 128;
	int blocksPerGrid = 8;
	FindKernel << <blocksPerGrid, threadsPerBlock >> > (d_A, a, size, d_found);

	hipDeviceSynchronize(); // Add synchronization
	hipMemcpy(&h_found, d_found, sizeof(bool), hipMemcpyDeviceToHost);
	hipFree(d_found);

	hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		std::cerr << "CUDA error: " << hipGetErrorString(error) << std::endl;
		return false;
	}

	return h_found;
}

int main()
{
	cout << "ExpIndex索引规模为1757" << endl;
	cout << "ExpIndex中最大文档ID为25205174,最小文档ID为0" << endl;
	cout << "ExpQuery最大查询规模为1000" << endl;
	int QueryNum;
	cout << "请输入问题规模(0-1000内的整型)：" << endl;

	cin >> QueryNum;
	LARGE_INTEGER frequency;

	LARGE_INTEGER start;
	LARGE_INTEGER end;
	LARGE_INTEGER end_f;
	double interval;
	// 获取性能计数器频率
	QueryPerformanceFrequency(&frequency);
	// 开始计时
	QueryPerformanceCounter(&start);
	ifstream readIndex("ExpIndex", ios::binary | ios::in);
	if (!readIndex) {
		cerr << "无法打开ExpIndex文件" << endl;
		return 1;
	}
	vector<vector<unsigned int>>index;
	unsigned int n; // 数组的长度
	int i = 0;
	while (readIndex.read(reinterpret_cast<char*>(&n), sizeof(n))) {
		vector<unsigned int> array(n); // 创建一个容器来存储数组元素
		if (!readIndex.read(reinterpret_cast<char*>(array.data()), n * sizeof(unsigned int)))
		{
			cerr << "读取ExpIndex第" << i << "数组失败" << endl;
		}
		sort(array.begin(), array.end());
		index.push_back(array);
		i++;
	}
	// 关闭文件流
	readIndex.close();
	ifstream readQuery("ExpQuery", ios::in);
	if (!readQuery) {
		cerr << "无法打开ExpQuery文件" << endl;
		return 1;
	}
	vector<vector<int>>query;
	string line;
	while (getline(readQuery, line))
	{
		vector<int> array; // 创建一个容器来存储数组元素
		stringstream readline(line);
		int a;
		while (readline >> a)
		{
			array.push_back(a);
		}
		sort(array.begin(), array.end());
		query.push_back(array);
	}
	// 关闭文件流
	readQuery.close();
	QueryPerformanceCounter(&end_f);
	interval = static_cast<double>(end_f.QuadPart - start.QuadPart) / frequency.QuadPart;

	std::cout << "读取文件时间: " << interval << " 秒" << std::endl;
	cout << "读取文件操作完成" << endl;

	//cout << "索引表的总长度" << index.size() << endl;
	vector<vector<unsigned int>>results;
	for (int i = 0; i < QueryNum; i++)
	{
		int num = query[i].size();
		vector<unsigned int>result;
		result = index[query[i][0]];
		for (int j = 1; j < num; j++)
		{
			unsigned int* d_index;
			hipMalloc((void**)&d_index, index[query[i][j]].size() * sizeof(unsigned int));
			hipMemcpy(d_index, index[query[i][j]].data(), index[query[i][j]].size() * sizeof(unsigned int), hipMemcpyHostToDevice);

			for (int k = 0; k < result.size(); k++)
			{
				if (!Find(d_index, result[k], index[query[i][j]].size()))
				{
					result.erase(result.begin() + k);
					k--;
				}
			}
			hipFree(d_index);
		}
		results.push_back(result);
	}
	// 结束计时
	QueryPerformanceCounter(&end);
	// 计算时间间隔
	interval = static_cast<double>(end.QuadPart - start.QuadPart) / frequency.QuadPart;

	std::cout << "代码执行时间: " << interval << " 秒" << std::endl;
	return 0;
}
