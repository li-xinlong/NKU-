#include "hip/hip_runtime.h"
#include<iostream>
#include<string>
#include<fstream>
#include<sstream>
#include<vector>
#include<algorithm>
#include<Windows.h>
#include<hip/hip_runtime.h>

using namespace std;

__global__ void FindKernel(const unsigned int* d_A, unsigned int a, bool* d_result, int size) {
	extern __shared__ unsigned int shared_A[];
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int tid = threadIdx.x;

	if (idx < size) {
		shared_A[tid] = d_A[idx];
	}
	__syncthreads();

	if (idx < size) {
		if (shared_A[tid] == a) {
			*d_result = true;
		}
	}
}

bool FindCUDA(vector<unsigned int>& A, unsigned int a) {
	unsigned int* d_A;
	bool* d_result;
	bool h_result = false;
	int size = A.size() * sizeof(unsigned int);

	hipMalloc((void**)&d_A, size);
	hipMalloc((void**)&d_result, sizeof(bool));

	hipMemcpy(d_A, A.data(), size, hipMemcpyHostToDevice);
	hipMemcpy(d_result, &h_result, sizeof(bool), hipMemcpyHostToDevice);

	int blockSize = 256;
	int numBlocks = (A.size() + blockSize - 1) / blockSize;
	int sharedMemSize = blockSize * sizeof(unsigned int);

	FindKernel << <numBlocks, blockSize, sharedMemSize >> > (d_A, a, d_result, A.size());

	hipMemcpy(&h_result, d_result, sizeof(bool), hipMemcpyDeviceToHost);

	hipFree(d_A);
	hipFree(d_result);

	return h_result;
}

int main() {
	cout << "ExpIndex索引规模为1757" << endl;
	cout << "ExpIndex中最大文档ID为25205174,最小文档ID为0" << endl;
	cout << "ExpQuery最大查询规模为1000" << endl;
	int QueryNum;
	cout << "请输入问题规模(0-1000内的整型)：" << endl;
	cin >> QueryNum;

	LARGE_INTEGER frequency, start, end, end_f;
	double interval;

	QueryPerformanceFrequency(&frequency);
	QueryPerformanceCounter(&start);

	ifstream readIndex("ExpIndex", ios::binary | ios::in);
	if (!readIndex) {
		cerr << "无法打开ExpIndex文件" << endl;
		return 1;
	}

	vector<vector<unsigned int>> index;
	unsigned int n;
	int i = 0;
	while (readIndex.read(reinterpret_cast<char*>(&n), sizeof(n))) {
		vector<unsigned int> array(n);
		if (!readIndex.read(reinterpret_cast<char*>(array.data()), n * sizeof(unsigned int))) {
			cerr << "读取ExpIndex第" << i << "数组失败" << endl;
		}
		sort(array.begin(), array.end());
		index.push_back(array);
		i++;
	}
	readIndex.close();

	ifstream readQuery("ExpQuery", ios::in);
	if (!readQuery) {
		cerr << "无法打开ExpQuery文件" << endl;
		return 1;
	}

	vector<vector<int>> query;
	string line;
	while (getline(readQuery, line)) {
		vector<int> array;
		stringstream readline(line);
		int a;
		while (readline >> a) {
			array.push_back(a);
		}
		sort(array.begin(), array.end());
		query.push_back(array);
	}
	readQuery.close();

	QueryPerformanceCounter(&end_f);
	interval = static_cast<double>(end_f.QuadPart - start.QuadPart) / frequency.QuadPart;
	cout << "读取文件时间: " << interval << " 秒" << endl;
	cout << "读取文件操作完成" << endl;

	vector<vector<unsigned int>> results;
	for (int i = 0; i < QueryNum; i++) {
		int num = query[i].size();
		vector<unsigned int> result;
		for (int j = 0; j < index[query[i][0]].size(); j++) {
			bool bo = true;
			unsigned int a = index[query[i][0]][j];
			for (int k = 1; k < num; k++) {
				if (!FindCUDA(index[query[i][k]], a)) {
					bo = false;
					break;
				}
			}
			if (bo) {
				result.push_back(a);
			}
		}
		results.push_back(result);
	}

	QueryPerformanceCounter(&end);
	interval = static_cast<double>(end.QuadPart - start.QuadPart) / frequency.QuadPart;
	cout << "代码执行时间: " << interval << " 秒" << endl;

	return 0;
}
